#include <iostream>
#include <complex>

#include <hipblas.h>
#include <hip/hip_runtime.h>

#include <ccglib/ccglib.hpp>

#include "config.h"

inline void cublas_check(hipblasStatus_t status) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    throw std::runtime_error(cublasGetStatusString(status));
  }
}

inline void cuda_check(hipError_t err) {
  if (err != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(err));
  }
}

int main() {
  Params<ccglib::float16, ccglib::float32, large> params;

  hipblasHandle_t handle;
  cublas_check(hipblasCreate(&handle));

  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;
  hipDataType Atype = HIP_C_32F;
  hipDataType Btype = HIP_C_32F;
  hipDataType Ctype = HIP_C_32F;
  hipblasComputeType_t Computetype = HIPBLAS_COMPUTE_32F_FAST_16F;

  hipblasGemmAlgo_t algo = CUBLAS_GEMM_DEFAULT_TENSOR_OP;

  int lda = params.K;
  int ldb = params.K;
  int ldc = params.N;

  std::complex<float> alpha = {1, 0};
  std::complex<float> beta = {1, 0};

  size_t bytes_a = params.M * params.K * sizeof(float) * 2;
  size_t bytes_b = params.N * params.K * sizeof(float) * 2;
  size_t bytes_c = params.M * params.N * sizeof(float) * 2;

  void *d_a;
  void *d_b;
  void *d_c;

  cuda_check(hipMalloc(&d_a, bytes_a));
  cuda_check(hipMalloc(&d_b, bytes_b));
  cuda_check(hipMalloc(&d_c, bytes_c));

  // should try _batched variant where the device pointers to a,b,c are arrays
  hipEvent_t start, end;
  cuda_check(hipEventCreate(&start));
  cuda_check(hipEventCreate(&end));

  cuda_check(hipEventRecord(start));
  cublas_check(hipblasGemmEx(handle, transa, transb, params.M, params.N, params.K,
                            &alpha, d_a, Atype, lda, d_b, Btype, ldb, &beta,
                            d_c, Ctype, ldc, Computetype, algo));

  cuda_check(hipEventRecord(end));
  cuda_check(hipEventSynchronize(end));
  cuda_check(hipDeviceSynchronize());

  float runtime;
  cuda_check(hipEventElapsedTime(&runtime, start, end));
  const double tflops = 8ULL * 1e-9 * params.M * params.N * params.K / runtime;

  std::cout << "Runtime: " << runtime << " ms" << std::endl;
  std::cout << "TFLOPS: " << tflops << std::endl;
  

  cublas_check(hipblasDestroy(handle));

  cuda_check(hipFree(d_a));
  cuda_check(hipFree(d_b));
  cuda_check(hipFree(d_c));
}

