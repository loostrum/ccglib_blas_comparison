#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>

#include <hipblaslt.h>
#include <hip/hip_runtime.h>

#include <ccglib/ccglib.hpp>

#include "config.h"

#define cublas_check(status) {cublas_assert((status), __FILE__, __LINE__);}
inline void cublas_assert(hipblasStatus_t status, const char *file, int line) {
  if (status != HIPBLAS_STATUS_SUCCESS) {
    std::string err = std::string(file) + " line " + std::to_string(line) + " " + std::string(cublasLtGetStatusString(status));
    throw std::runtime_error(err);
  }
}

inline void cuda_check(hipError_t err) {
  if (err != hipSuccess) {
    throw std::runtime_error(hipGetErrorString(err));
  }
}

int main() {
  Params<ccglib::float16, ccglib::float32, large> params;

  hipblasLtHandle_t handle;
  cublas_check(hipblasLtCreate(&handle));

  hipComplex alpha = {1, 0};
  hipComplex beta = {0, 0};

  hipblasOperation_t transa = HIPBLAS_OP_T;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  // create matrix multiplication descriptor
  hipblasLtMatmulDesc_t desc;
  // compute type (= effectively if/which tensor cores are used), scale type (=alpha,beta)
  cublas_check(hipblasLtMatmulDescCreate(&desc, HIPBLAS_COMPUTE_32F, HIP_C_32F));
  cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  cublas_check(hipblasLtMatmulDescSetAttribute(desc, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));


  // planar layout, cublas needs offset between real and imag part
  long offset_a = params.M * params.K * sizeof(half);
  long offset_b = params.N * params.K * sizeof(half);
  long offset_c = params.M * params.N * sizeof(float);

  // create layout descriptors
  hipblasLtMatrixLayout_t layout_a, layout_b, layout_c;
  // type, rows, cols, leading dimension (col major: nr elements from one col to the next)
  cublas_check(hipblasLtMatrixLayoutCreate(&layout_a, HIP_C_16F, params.M, params.K, params.K));
  cublas_check(hipblasLtMatrixLayoutCreate(&layout_b, HIP_C_16F, params.K, params.N, params.K));
  cublas_check(hipblasLtMatrixLayoutCreate(&layout_c, HIP_C_32F, params.M, params.N, params.N));

  // setting the plane offset to a nonzero value makes this a planar mode (vs interleaved real,imag)
  cublas_check(hipblasLtMatrixLayoutSetAttribute(layout_a, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &offset_a, sizeof(offset_a)));
  cublas_check(hipblasLtMatrixLayoutSetAttribute(layout_b, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &offset_b, sizeof(offset_b)));
  cublas_check(hipblasLtMatrixLayoutSetAttribute(layout_c, CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET, &offset_c, sizeof(offset_c)));

  // create matrices
  size_t bytes_a = params.M * params.K * sizeof(half) * 2;
  size_t bytes_b = params.N * params.K * sizeof(half) * 2;
  size_t bytes_c = params.M * params.N * sizeof(float) * 2;

  __half *d_a;
  __half *d_b;
  float *d_c;

  cuda_check(hipMalloc(&d_a, bytes_a));
  cuda_check(hipMalloc(&d_b, bytes_b));
  cuda_check(hipMalloc(&d_c, bytes_c));


  // launch
  hipEvent_t start, end;
  cuda_check(hipEventCreate(&start));
  cuda_check(hipEventCreate(&end));

  cuda_check(hipEventRecord(start));
  cublas_check(hipblasLtMatmul(handle,
                              desc,
                              &alpha,
                              d_a,
                              layout_a,
                              d_b,
                              layout_b,
                              &beta,
                              d_c,
                              layout_c,
                              d_c,
                              layout_c,
                              NULL, // algo
                              NULL, // workspace
                              0, // workspace size in bytes
                              0 // stream
                              ));
  cuda_check(hipEventRecord(end));
  cuda_check(hipEventSynchronize(end));
  cuda_check(hipDeviceSynchronize());

  float runtime;
  cuda_check(hipEventElapsedTime(&runtime, start, end));
  const double tflops = 8ULL * 1e-9 * params.M * params.N * params.K / runtime;

  std::cout << "Runtime: " << runtime << " ms" << std::endl;
  std::cout << "TFLOPS: " << tflops << std::endl;
  
  cublas_check(hipblasLtMatrixLayoutDestroy(layout_a));
  cublas_check(hipblasLtMatrixLayoutDestroy(layout_b));
  cublas_check(hipblasLtMatrixLayoutDestroy(layout_c));
  cublas_check(hipblasLtMatmulDescDestroy(desc));

  cublas_check(hipblasLtDestroy(handle));

  cuda_check(hipFree(d_a));
  cuda_check(hipFree(d_b));
  cuda_check(hipFree(d_c));
}

